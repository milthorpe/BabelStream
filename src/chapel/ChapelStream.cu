#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

void check_error(void)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}
extern "C" {
int get_device_driver_version(const int device) {
    hipSetDevice(device);
    check_error();
    int driver;
    hipDriverGetVersion(&driver);
    check_error();
    return driver;
}

std::string get_device_name(const int device) {
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  check_error();
  std::string device_name(props.name);
  return device_name;
}
}